#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include "Utilities.h"

#include "BinaryVoxelMatrix.h"
#include "BinaryMesh.h"
#include "ImportBinaryMeshWorkflow.h"

#include <iostream>
#include <string>
#include <fstream>






struct CUDA_VoxelMatrix
{
	voxel * raw_voxel_store;

	float3 * cuda_voxels_start;
	float3 * cuda_voxels_end;

	int * cuda_voxel_tags;

	//hipExtent cuda_matrix_extent;
	//hipChannelFormatDesc cuda_voxel_format_desc;

	int3 resolution;
	float3 start, end;

	CUDA_VoxelMatrix( int res_width, int res_height, int res_depth, float3 bounds_start, float3 bounds_end  )
	{
		resolution = make_int3( res_width, res_height, res_depth );

		//cuda_matrix_extent = make_hipExtent( width, height, depth );
		//cuda_format_desc = hipCreateChannelDesc(  )
		//cuda_voxel_format_desc = hipCreateChannelDesc( 32, )

		start = bounds_start;
		end = bounds_end;

		float3 stride = (end - start) / resolution;



		int numVoxels = res_width * res_height * res_depth;
		hipMalloc( &cuda_voxels_start, numVoxels * sizeof( float ) *  3 );
		hipMalloc( &cuda_voxels_end, numVoxels * sizeof( float ) * 3 );
		hipMalloc( &cuda_voxel_tags, numVoxels * sizeof( int ) );

		raw_voxel_store = new voxel[numVoxels];

		for( int z = 0, idx = 0; z < res_depth; z++ )
		{
			for( int y = 0; y < res_height; y++ )
			{
				for( int x = 0; x < res_width; x++ )
				{
					voxel current;
					current.start.x = start.x + stride.x * x;
					current.start.y = start.y + stride.y * y;
					current.start.z = start.z + stride.z * z;

					current.end.x = start.x + stride.x * (x + 1);
					current.end.y = start.y + stride.y * (y + 1);
					current.end.z = start.z + stride.z * (z + 1);

					raw_voxel_store[idx++] = current;
				}
			}
		}

		float * voxel_starts = new float[numVoxels * 3]; // 3 floats per voxel
		for( int i = 0; i < numVoxels; i++ )
		{
			const auto & voxel = raw_voxel_store[i];
			voxel_starts[i * 3 + 0] = voxel.start.x;
			voxel_starts[i * 3 + 1] = voxel.start.y;
			voxel_starts[i * 3 + 2] = voxel.start.z;
		}
		hipMemcpy( cuda_voxels_start, voxel_starts, numVoxels * 3 * sizeof( float ), hipMemcpyHostToDevice );
		delete[] voxel_starts;

		float * voxel_ends = new float[numVoxels * 3]; // 3 floats per voxel
		for( int i = 0; i < numVoxels; i++ )
		{
			const auto & voxel = raw_voxel_store[i];
			voxel_ends[i * 3 + 0] = voxel.end.x;
			voxel_ends[i * 3 + 1] = voxel.end.y;
			voxel_ends[i * 3 + 2] = voxel.end.z;
		}
		hipMemcpy( cuda_voxels_end, voxel_ends, numVoxels * 3 * sizeof( float ), hipMemcpyHostToDevice );
		delete[] voxel_ends;

		int * voxel_tags = new int[numVoxels];
		for( int i = 0; i < numVoxels; i++ )
			voxel_tags[i] = 0;
		hipMemcpy( cuda_voxel_tags, voxel_tags, numVoxels * sizeof( int ), hipMemcpyHostToDevice );
		delete[] voxel_tags;
	}

	~CUDA_VoxelMatrix( )
	{
		hipFree( cuda_voxels_start );
		hipFree( cuda_voxels_end );
		hipFree( cuda_voxel_tags );
	}

	void CopyTo( voxel * voxels_store, int num_voxels )
	{
		NOT_YET_IMPLEMENTED( );
	}
};



#define TRIS_PER_TILE 128
#define TILE_SIZE 128

__global__ void tagVoxelsKernel(
	int * targetTags,
	const float3 * voxel_starts, const float3 * voxel_ends,
	const float3 * chunk_tris, int num_chunk_tris,
	int3 matrix_resolution,
	int3 voxel_offset // Offset of this span within the matrix
	)
{
	__shared__ float3 current_tile_tri_centers[TRIS_PER_TILE];

	int numTileTris = min( num_chunk_tris - blockIdx.x * TILE_SIZE, TRIS_PER_TILE );
	
	int threadStride = TILE_SIZE;
	int numIterations = (int)ceil( (float)TRIS_PER_TILE / (float)TILE_SIZE );
	int trisOffset = blockIdx.x * TRIS_PER_TILE;
	for( int i = 0; i < numIterations; i++ )
	{
		int bidx = i * threadStride + threadIdx.x;
		float3 a = chunk_tris[trisOffset + bidx + 0];
		float3 b = chunk_tris[trisOffset + bidx + 1];
		float3 c = chunk_tris[trisOffset + bidx + 2];

		float3 center;
		center.x = 0.333333f * ( a.x + b.x + c.x );
		center.y = 0.333333f * ( a.y + b.y + c.y );
		center.z = 0.333333f * ( a.z + b.z + c.z );

		current_tile_tri_centers[bidx] = center;
	}

	__syncthreads( );

	int vidx = threadIdx.x * blockDim.y * blockDim.z + threadIdx.y * blockDim.z + threadIdx.z;

	float3 vstart = voxel_starts[vidx];
	float3 vend = voxel_ends[vidx];

	int numTags = 0;
	for( int i = 0; i < numTileTris; i++ )
	{
		float3 c = current_tile_tri_centers[i];
		if( vstart.x < c.x &&
			vstart.y < c.y &&
			vstart.z < c.z &&
			vend.x > c.x &&
			vend.y > c.y &&
			vend.z > c.z )
			++numTags;
	}

	atomicAdd( targetTags + vidx, numTags );
}

int main()
{
	std::vector<mesh_chunk> * mesh_set;
	workflow_import_binary_mesh_set( "../QuickData/patients/bs/bsArteries/surfaces", &mesh_set );

	int numChunks = mesh_set->size( );
	std::vector<float3 *> mesh_set_tris;
	std::vector<int> mesh_set_tris_counts;
	for( int i = 0; i < numChunks; i++ )
	{
		const auto & chunk = mesh_set->at( i );
		float3 * cudaStore;
		hipMalloc( &cudaStore, chunk.num_tris * sizeof(float3) * 3 );
		hipMemcpy( cudaStore, chunk.tris.data( ), sizeof( float3 ) * 3, hipMemcpyHostToDevice );

		mesh_set_tris.push_back( cudaStore );
		mesh_set_tris_counts.push_back( chunk.num_tris );
	}

	MeshPartitionScheme mesh_partition_scheme;
	mesh_partition_scheme.LoadFrom( "../QuickData/patients/bs/bsArteries/surfaces/partitions.binmeshscheme" );


	//	The overlapped regions between chunk and voxels will always be a box, definable by start and end points.
	std::vector<int3> chunk_voxel_overlap_start;
	std::vector<int3> chunk_voxel_overlap_end;

	std::cout << "Generating voxel-chunk bounds... ";
	CUDA_VoxelMatrix vm( 512, 512, 512, make_float3( -30.0f ), make_float3( 190.0f ) );
	int3 vm_dims = vm.resolution;

	auto vmStart = vm.start;
	auto stride = (vm.end - vm.start) / vm.resolution;
	auto invStride = make_float3( 1.0f ) / stride;

	std::cout << std::endl;
	//	For every chunk
	for( int i = 0; i < mesh_set->size( ); i++ )
	{
		auto & chunk = mesh_set->at( i );
		auto desc = chunk.bounds;


		float3 transformed_min = (desc.bounds_start - vmStart) * invStride;
		float3 transformed_max = (desc.bounds_end - vmStart) * invStride;

		int3 overlap_start = make_int3(
			floorf( transformed_min.x ),
			floorf( transformed_min.y ),
			floorf( transformed_min.z )
			);

		int3 overlap_end = make_int3(
			ceilf( transformed_max.x ),
			ceilf( transformed_max.y ),
			ceilf( transformed_max.z )
			);

		if( overlap_start.x < 0 || overlap_start.y < 0 || overlap_start.z < 0 ||
			overlap_end.x < 0 || overlap_end.y < 0 || overlap_end.z < 0 ||
			overlap_start.x >= vm_dims.x || overlap_start.y >= vm_dims.y || overlap_start.z >= vm_dims.z ||
			overlap_end.x >= vm_dims.x || overlap_end.y > vm_dims.y || vm_dims.z > vm_dims.z )
		{
			std::cout << "VOXEL MATRIX SPACE CANNOT CONTAIN MESH" << std::endl;
			NOT_YET_IMPLEMENTED( );
		}

		chunk_voxel_overlap_start.push_back( overlap_start );
		chunk_voxel_overlap_end.push_back( overlap_end );
	}

	
	
	for( int c = 0; c < numChunks; c++ )
	{
		std::cout << "c: " << c << std::endl;
		auto voxels_start = chunk_voxel_overlap_start[c];
		auto voxels_end = chunk_voxel_overlap_end[c];

		const auto & chunk = mesh_set->at( c );
		int numBlocks = (int)ceil( (float)chunk.num_tris / (float)TRIS_PER_TILE );
		dim3 voxelExtents( voxels_end.x - voxels_start.x, voxels_end.y - voxels_start.y, voxels_end.z - voxels_start.z );

		tagVoxelsKernel<<<numBlocks, voxelExtents>>>(
					vm.cuda_voxel_tags,
					vm.cuda_voxels_start, vm.cuda_voxels_end,
					mesh_set_tris[c],
					chunk.num_tris,
					vm.resolution,
					voxels_start
					);

		auto result = hipDeviceSynchronize( );
		std::cout << "sync: " << result << std::endl;
		Sleep( 1 );
	}

	int * result_tags = new int[vm.resolution.x * vm.resolution.y * vm.resolution.z];
	hipMemcpy( result_tags, vm.cuda_voxel_tags, sizeof( int ) * vm.resolution.x * vm.resolution.y * vm.resolution.z, hipMemcpyDeviceToHost );
	for( int i = 0; i < vm.resolution.x * vm.resolution.y * vm.resolution.z; i++ )
		std::cout << result_tags[i] << ", ";

	std::string j;
	std::getline( std::cin, j );

    // Add vectors in parallel.
	/*
    hipError_t cudaStatus = addWithCuda_Syncronous(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }
	

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
	*/

    return 0;
}
